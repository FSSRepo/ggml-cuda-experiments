#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <torch/library.h>
#include <cmath>
#include <vector>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "sputnik/vector_utils.h"

namespace {

template <typename integer>
constexpr __host__ __device__ inline integer ceil_div(integer n, integer m) {
  return (n + m - 1) / m;
}

template <typename scalar_t>
constexpr __host__ __device__ bool integerIsPowerOf2(scalar_t v) {
  return (v && !(v & (v - 1)));
}

template <typename scalar_t>
__device__ __forceinline__ void iMul(scalar_t x1, float4* out) {
  out[0].x *= x1;
  out[0].y *= x1;
  out[0].z *= x1;
  out[0].w *= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iMul(scalar_t x1, float2* out) {
  out[0].x *= x1;
  out[0].y *= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iMul(scalar_t x1, float* out) {
  out[0] *= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iDiv(scalar_t x1, float4* out) {
  out[0].x /= x1;
  out[0].y /= x1;
  out[0].z /= x1;
  out[0].w /= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iDiv(scalar_t x1, float2* out) {
  out[0].x /= x1;
  out[0].y /= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iDiv(scalar_t x1, float* out) {
  out[0] /= x1;
}

template <typename scalar_t, int WARP_SIZE>
__device__ __forceinline__ scalar_t warpSum(scalar_t val) {
  for (int stride = WARP_SIZE / 2; stride > 0; stride >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, stride, WARP_SIZE);
  }
  return val;
}

template <typename scalar_t, int WARP_SIZE>
__device__ __forceinline__ float2 warpSum(float2 val) {
  for (int stride = WARP_SIZE / 2; stride > 0; stride >>= 1) {
    val.x += __shfl_xor_sync(0xffffffff, val.x, stride, WARP_SIZE);
    val.y += __shfl_xor_sync(0xffffffff, val.y, stride, WARP_SIZE);
  }
  return val;
}

template <typename scalar_t, int WARP_SIZE>
__device__ __forceinline__ float4 warpSum(float4 val) {
  for (int stride = WARP_SIZE / 2; stride > 0; stride >>= 1) {
    val.x += __shfl_xor_sync(0xffffffff, val.x, stride, WARP_SIZE);
    val.y += __shfl_xor_sync(0xffffffff, val.y, stride, WARP_SIZE);
    val.z += __shfl_xor_sync(0xffffffff, val.z, stride, WARP_SIZE);
    val.w += __shfl_xor_sync(0xffffffff, val.w, stride, WARP_SIZE);
  }
  return val;
}

template <typename scalar_t, int WARP_SIZE>
__device__ __forceinline__ scalar_t warpMax(scalar_t val) {
  for (int stride = WARP_SIZE / 2; stride > 0; stride >>= 1) {
    scalar_t tmp = __shfl_xor_sync(0xffffffff, val, stride, WARP_SIZE);
    val = tmp > val ? tmp : val;
  }
  return val;
}

template <typename scalar_t, typename vec_t, int kBlockSizeK, int kBlockSizeQ>
__device__ void compute_dot(
    vec_t* queries[kBlockSizeQ],
    vec_t* keys,
    scalar_t out[kBlockSizeQ][kBlockSizeK],
    int64_t K) {
  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);
  scalar_t scale = 1.0 / std::sqrt(scalar_t(K));
  vec_t q_i[kBlockSizeQ];
  for (int64_t k = 0; k < K / kVecSize; k += 1) {
#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
      q_i[q_item_idx] = __ldg(queries[q_item_idx] + k);
      iMul(scale, q_i + q_item_idx);
    }
#pragma unroll
    for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
      vec_t k_i = keys[k + K / kVecSize * k_item_idx];
#pragma unroll
      for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
        sputnik::VectorCompute<vec_t>::Dot(
            q_i[q_item_idx], k_i, &out[q_item_idx][k_item_idx]);
      }
    }
  }
}

template <
    typename scalar_t,
    typename vec_t,
    int kBlockSizeK,
    int kBlockSizeQ,
    int BUFFER_SIZE>
__device__ void compute_final_mult(
    vec_t* vi,
    scalar_t s_delta[kBlockSizeQ][kBlockSizeK],
    scalar_t m_delta[kBlockSizeQ],
    vec_t buffer[kBlockSizeQ][BUFFER_SIZE] /*TODO [BUFFER_SIZE limitation]*/,
    int64_t K) {
  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);

  for (int64_t k = 0; k < K / kVecSize; k += 1) {
#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
      iMul<scalar_t>(m_delta[q_item_idx], &buffer[q_item_idx][k]);
    }
#pragma unroll
    for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
      vec_t tmp2 = vi[k + K / kVecSize * k_item_idx];

#pragma unroll
      for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
        sputnik::VectorCompute<vec_t>::FMA(
            s_delta[q_item_idx][k_item_idx], tmp2, &buffer[q_item_idx][k]);
      }
    }
  }
}

template <typename scalar_t, int kBlockSizeK, int kBlockSizeQ>
__device__ __forceinline__ void compute_max(
    scalar_t a[kBlockSizeQ][kBlockSizeK],
    scalar_t b[kBlockSizeQ],
    scalar_t out[kBlockSizeQ]) {
#pragma unroll
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    out[q_item_idx] =
        a[q_item_idx][0] > b[q_item_idx] ? a[q_item_idx][0] : b[q_item_idx];
#pragma unroll
    for (int64_t k_item_idx = 1; k_item_idx < kBlockSizeK; k_item_idx++) {
      out[q_item_idx] = a[q_item_idx][k_item_idx] > out[q_item_idx]
          ? a[q_item_idx][k_item_idx]
          : out[q_item_idx];
    }
  }
}

template <typename scalar_t, int kBlockSizeK, int kBlockSizeQ>
__device__ __forceinline__ void compute_scaling_coeffs(
    scalar_t m_i[kBlockSizeQ],
    scalar_t m_prime[kBlockSizeQ],
    scalar_t si[kBlockSizeQ][kBlockSizeK],
    scalar_t m_delta[kBlockSizeQ],
    scalar_t s_delta[kBlockSizeQ][kBlockSizeK]) {
#pragma unroll
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++)
    m_delta[q_item_idx] = std::exp(m_prime[q_item_idx] - m_i[q_item_idx]);
#pragma unroll
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++)
#pragma unroll
    for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++)
      s_delta[q_item_idx][k_item_idx] =
          std::exp(si[q_item_idx][k_item_idx] - m_i[q_item_idx]);
}

template <typename scalar_t, int kBlockSizeK, int kBlockSizeQ>
__device__ __forceinline__ void update_scaling_coeffs(
    scalar_t m_delta[kBlockSizeQ],
    scalar_t m_i[kBlockSizeQ],
    scalar_t s_delta[kBlockSizeQ][kBlockSizeK],
    scalar_t m_prime[kBlockSizeQ],
    scalar_t s_prime[kBlockSizeQ]) {
#pragma unroll
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    s_prime[q_item_idx] = s_prime[q_item_idx] * m_delta[q_item_idx];
#pragma unroll
    for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++)
      s_prime[q_item_idx] += s_delta[q_item_idx][k_item_idx];

    m_prime[q_item_idx] = m_i[q_item_idx];
  }
}

template <
    typename scalar_t,
    typename vec_t,
    int kBlockSizeK,
    int kBlockSizeQ,
    int BUFFER_SIZE>
__device__ void compute_loop(
    vec_t* query_block[kBlockSizeQ],
    vec_t* key_i,
    vec_t* value_i,
    scalar_t m_prime[kBlockSizeQ],
    scalar_t s_prime[kBlockSizeQ],
    vec_t buffer[kBlockSizeQ][BUFFER_SIZE] /*TODO [BUFFER_SIZE limitation]*/,
    int64_t K) {
  scalar_t si[kBlockSizeQ][kBlockSizeK] = {0};
  compute_dot<scalar_t, vec_t, kBlockSizeK, kBlockSizeQ>(
      query_block, key_i, si, K);

  scalar_t m_i[kBlockSizeQ];
  compute_max<scalar_t, kBlockSizeK, kBlockSizeQ>(si, m_prime, m_i);

  scalar_t m_delta[kBlockSizeQ];
  scalar_t s_delta[kBlockSizeQ][kBlockSizeK];

  compute_scaling_coeffs<scalar_t, kBlockSizeK, kBlockSizeQ>(
      m_i, m_prime, si, m_delta, s_delta);

  compute_final_mult<scalar_t, vec_t, kBlockSizeK, kBlockSizeQ, BUFFER_SIZE>(
      value_i, s_delta, m_delta, buffer, K);

  update_scaling_coeffs<scalar_t, kBlockSizeK, kBlockSizeQ>(
      m_delta, m_i, s_delta, m_prime, s_prime);
}

template <
    typename scalar_t,
    typename vec_t,
    int kBlockSizeQ,
    int WARP_SIZE,
    int BUFFER_SIZE>
__device__ __forceinline__ void aggregate_coeffs(
    scalar_t m_prime[kBlockSizeQ],
    scalar_t s_prime[kBlockSizeQ],
    vec_t buffer[kBlockSizeQ][BUFFER_SIZE] /*TODO [BUFFER_SIZE limitation]*/,
    int64_t K) {
  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    scalar_t m_i = m_prime[q_item_idx];
    scalar_t s_i = s_prime[q_item_idx];
    m_prime[q_item_idx] = warpMax<scalar_t, WARP_SIZE>(m_prime[q_item_idx]);
    scalar_t m_delta = std::exp(m_i - m_prime[q_item_idx]);
    scalar_t s_delta = s_i * m_delta;
    s_delta = warpSum<scalar_t, WARP_SIZE>(s_delta);
    s_prime[q_item_idx] = s_delta;
    for (int64_t k = 0; k < K / kVecSize; k += 1) {
      vec_t tmp = buffer[q_item_idx][k];
      iMul<scalar_t>(m_delta, &tmp);
      tmp = warpSum<vec_t, WARP_SIZE>(tmp);
      buffer[q_item_idx][k] = tmp;
    }
  }
}

template <
    bool first,
    typename scalar_t,
    typename vec_t,
    int kBlockSizeK,
    int kBlockSizeQ,
    int BUFFER_SIZE,
    int WARP_SIZE>
struct UnrollLoop {
  static __device__ __forceinline__ void eval(
      vec_t* query_block[kBlockSizeQ],
      at::TensorAccessor<scalar_t, 2> key,
      at::TensorAccessor<scalar_t, 2> value,
      scalar_t m_prime[kBlockSizeQ],
      scalar_t s_prime[kBlockSizeQ],
      vec_t buffer[kBlockSizeQ][BUFFER_SIZE] /*TODO [BUFFER_SIZE limitation]*/,
      int64_t K,
      int64_t N) {
    constexpr int64_t step = kBlockSizeK * WARP_SIZE;
    int64_t l;
    if (first) {
      l = threadIdx.x * kBlockSizeK;
    } else {
      l = N - (N & (2 * step - 1)) + threadIdx.x * kBlockSizeK;
    }
    // this is equivalent to N - N % step, but faster
    // guaranteed to be the same as step is a power of 2
    int64_t end_iter = N - (N & (step - 1));
    // if (l < end_iter) {
    {
      for (; l < end_iter; l += step) {
        auto key_i = reinterpret_cast<vec_t*>(key[l].data());
        auto value_i = reinterpret_cast<vec_t*>(value[l].data());

        compute_loop<scalar_t, vec_t, kBlockSizeK, kBlockSizeQ, BUFFER_SIZE>(
            query_block, key_i, value_i, m_prime, s_prime, buffer, K);
      }
    }
    {
      UnrollLoop<
          false,
          scalar_t,
          vec_t,
          kBlockSizeK / 2,
          kBlockSizeQ,
          BUFFER_SIZE,
          WARP_SIZE>::
          eval(query_block, key, value, m_prime, s_prime, buffer, K, N);
    }
  }
};

template <
    bool first,
    typename scalar_t,
    typename vec_t,
    int kBlockSizeQ,
    int BUFFER_SIZE,
    int WARP_SIZE>
struct UnrollLoop<
    first,
    scalar_t,
    vec_t,
    0,
    kBlockSizeQ,
    BUFFER_SIZE,
    WARP_SIZE> {
  static __device__ __forceinline__ void eval(
      vec_t* query_block[kBlockSizeQ],
      at::TensorAccessor<scalar_t, 2> key,
      at::TensorAccessor<scalar_t, 2> value,
      scalar_t m_prime[kBlockSizeQ],
      scalar_t s_prime[kBlockSizeQ],
      vec_t buffer[kBlockSizeQ][BUFFER_SIZE] /*TODO [BUFFER_SIZE limitation]*/,
      int64_t K,
      int64_t N) {}
};

template <
    typename scalar_t,
    typename vec_t,
    int kBlockSizeK,
    int kBlockSizeQ,
    int WARP_SIZE,
    int BUFFER_SIZE>
__global__ void attention_kernel(
    at::PackedTensorAccessor<scalar_t, 3> output,
    at::PackedTensorAccessor<scalar_t, 3> query,
    at::PackedTensorAccessor<scalar_t, 3> key,
    at::PackedTensorAccessor<scalar_t, 3> value) {
  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);
  static_assert(
      integerIsPowerOf2(kBlockSizeK * WARP_SIZE),
      "kBlockSizeK * WARP_SIZE should be a power of 2");
  int64_t K = query.size(2);
  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);

  int64_t batch_idx = blockIdx.y;
  int64_t query_idx =
      blockIdx.x * (blockDim.y * kBlockSizeQ) + threadIdx.y * kBlockSizeQ;

  if (query_idx >= M)
    return;

  vec_t* query_block[kBlockSizeQ];
  vec_t* output_block[kBlockSizeQ];
  // TODO [BUFFER_SIZE limitation]: the current strategy assumes a
  // statically-known size for K. Ideally we would like to remove this
  // limitation in the future, so that any K is supported
  vec_t buffer[kBlockSizeQ][BUFFER_SIZE] = {0};
  scalar_t s_prime[kBlockSizeQ] = {0};
  scalar_t m_prime[kBlockSizeQ];
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    int64_t index = query_idx + q_item_idx;
    index = index >= M ? M - 1 : index;
    query_block[q_item_idx] =
        reinterpret_cast<vec_t*>(query[batch_idx][index].data());
    output_block[q_item_idx] =
        reinterpret_cast<vec_t*>(output[batch_idx][index].data());
    m_prime[q_item_idx] = -std::numeric_limits<scalar_t>::infinity();
  }
#if 0
  // this for now makes things slower
  UnrollLoop<true, scalar_t, vec_t, kBlockSizeK, kBlockSizeQ, BUFFER_SIZE, WARP_SIZE>::eval(query_block, key[batch_idx], value[batch_idx], m_prime, s_prime, buffer, K, N);
#else
  int64_t l = threadIdx.x * kBlockSizeK;
  constexpr int64_t step = kBlockSizeK * WARP_SIZE;
  // this is equivalent to N - N % step, but faster
  // guaranteed to be the same as step is a power of 2
  int64_t end_iter = N - (N & (step - 1));
  for (; l < end_iter; l += step) {
    auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
    auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());

    compute_loop<scalar_t, vec_t, kBlockSizeK, kBlockSizeQ, BUFFER_SIZE>(
        query_block, key_i, value_i, m_prime, s_prime, buffer, K);
  }

  {
    // TODO: unroll this in a generic manner
    l = N - (N & (step - 1)) + threadIdx.x * (kBlockSizeK / 2);
    end_iter = N - (N & (step / 2 - 1));
    for (; l < end_iter; l += step / 2) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, kBlockSizeK / 2, kBlockSizeQ, BUFFER_SIZE>(
          query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }

    l = N - (N & (step / 2 - 1)) + threadIdx.x * (kBlockSizeK / 4);
    end_iter = N - (N & (step / 4 - 1));
    for (; l < end_iter; l += step / 4) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, kBlockSizeK / 4, kBlockSizeQ, BUFFER_SIZE>(
          query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }

    l = N - (N & (step / 4 - 1)) + threadIdx.x * (kBlockSizeK / 8);
    end_iter = N - (N & (step / 8 - 1));
    for (; l < end_iter; l += step / 8) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, kBlockSizeK / 8, kBlockSizeQ, BUFFER_SIZE>(
          query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }

    l = N - (N & (step / 8 - 1)) + threadIdx.x * (kBlockSizeK / 16);
    end_iter = N - (N & (step / 16 - 1));
    for (; l < end_iter; l += step / 16) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, kBlockSizeK / 16, kBlockSizeQ, BUFFER_SIZE>(
          query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }

    l = N - (N & (step / 16 - 1)) + threadIdx.x;
    for (; l < N; l += blockDim.x) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, 1, kBlockSizeQ, BUFFER_SIZE>(
          query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }
  }
#endif

  aggregate_coeffs<scalar_t, vec_t, kBlockSizeQ, WARP_SIZE, BUFFER_SIZE>(
      m_prime, s_prime, buffer, K);

  for (int64_t k = threadIdx.x; k < K / kVecSize; k += blockDim.x) {
    vec_t tmp;

#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
      tmp = buffer[q_item_idx][k];
      iDiv<scalar_t>(s_prime[q_item_idx], &tmp);

      output_block[q_item_idx][k] = tmp;
    }
  }
}

at::Tensor attention(
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value
    // const at::Tensor& mask
) {
  TORCH_CHECK(query.dim() == key.dim());
  TORCH_CHECK(query.dim() == value.dim());
  // TORCH_CHECK(query.dim() == mask.dim());
  TORCH_CHECK(query.dim() == 3);
  TORCH_CHECK(query.size(2) == key.size(2));
  TORCH_CHECK(query.size(0) == key.size(0));

  TORCH_CHECK(query.size(0) == value.size(0));
  TORCH_CHECK(key.size(1) == value.size(1));
  TORCH_CHECK(
      query.size(2) ==
      value.size(2)); // TODO: drop this limitation in the future

  TORCH_CHECK(query.is_cuda(), "query must be a CUDA tensor");
  TORCH_CHECK(key.is_cuda(), "key must be a CUDA tensor");
  TORCH_CHECK(value.is_cuda(), "value must be a CUDA tensor");

  TORCH_CHECK(!query.is_sparse(), "query must be a dense tensor");
  TORCH_CHECK(!key.is_sparse(), "key must be a dense tensor");
  TORCH_CHECK(!value.is_sparse(), "value must be a dense tensor");

  // TODO drop this limitation in the future
  TORCH_CHECK(query.is_contiguous());
  TORCH_CHECK(key.is_contiguous());
  TORCH_CHECK(value.is_contiguous());

  // TODO: support other dtypes in the future
  TORCH_CHECK(
      query.scalar_type() == at::ScalarType::Float,
      "Only float32 type is supported for now");

  at::cuda::CUDAGuard device_guard(query.device());

  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);
  int64_t K = query.size(2);

  at::Tensor res = at::zeros({B, M, K}, query.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  constexpr int WARP_SIZE = 4;

  constexpr int kBlockSizeK = 32;
  constexpr int kBlockSizeQ = 2;

  constexpr int TILE_SIZE = 32;
  constexpr int BUFFER_SIZE = 8;

  dim3 grid(ceil_div(M, int64_t(TILE_SIZE)), B);
  dim3 block(WARP_SIZE, TILE_SIZE / kBlockSizeQ);

  using scalar_t = float;

  if ((K % 4) == 0) {
    TORCH_CHECK(
        K / 4 <= BUFFER_SIZE,
        "For now only a certain number of K values are supported. Let us know if you hit this and we will fix it");
    attention_kernel<
        scalar_t,
        float4,
        kBlockSizeK,
        kBlockSizeQ,
        WARP_SIZE,
        BUFFER_SIZE><<<grid, block, 0, stream>>>(
        res.packed_accessor<scalar_t, 3>(),
        query.packed_accessor<scalar_t, 3>(),
        key.packed_accessor<scalar_t, 3>(),
        value.packed_accessor<scalar_t, 3>());
  } else if ((K % 2) == 0) {
    TORCH_CHECK(
        K / 2 <= BUFFER_SIZE,
        "For now only a certain number of K values are supported. Let us know if you hit this and we will fix it");
    attention_kernel<
        scalar_t,
        float2,
        kBlockSizeK,
        kBlockSizeQ,
        WARP_SIZE,
        BUFFER_SIZE><<<grid, block, 0, stream>>>(
        res.packed_accessor<scalar_t, 3>(),
        query.packed_accessor<scalar_t, 3>(),
        key.packed_accessor<scalar_t, 3>(),
        value.packed_accessor<scalar_t, 3>());

  } else {
    TORCH_CHECK(
        K <= BUFFER_SIZE,
        "For now only a certain number of K values are supported. Let us know if you hit this and we will fix it");
    attention_kernel<
        scalar_t,
        float,
        kBlockSizeK,
        kBlockSizeQ,
        WARP_SIZE,
        BUFFER_SIZE><<<grid, block, 0, stream>>>(
        res.packed_accessor<scalar_t, 3>(),
        query.packed_accessor<scalar_t, 3>(),
        key.packed_accessor<scalar_t, 3>(),
        value.packed_accessor<scalar_t, 3>());
  }

  return res;
}

} // namespace

TORCH_LIBRARY_IMPL(xformers, CUDA, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("xformers::efficient_attention"),
      TORCH_FN(attention));
}
