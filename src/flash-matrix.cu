#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>
#include "cuda_info.h"
#include "utils.h"

#define WARP_SIZE 32
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

typedef nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, nvcuda::wmma::row_major> MatrixA;
typedef nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, nvcuda::wmma::col_major> MatrixBT;
typedef nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, nvcuda::wmma::row_major> MatrixB;
typedef nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float>                          Accum;

static __device__ __forceinline__ float warp_reduce_max(float x) {
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        x = fmaxf(__shfl_xor_sync(0xffffffff, x, mask, 32), x);
    }
    return x;
}

static __device__ __forceinline__ float warp_reduce_sum(float x) {
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        x += __shfl_xor_sync(0xffffffff, x, mask, 32);
    }
    return x;
}

#define FA_KV_BLOCK_256

template<int head_dim, int num_warps, int kv_tensor, int kv_block>
__global__ void flash_attn(const half* query,
    half* key /* reuse key buffer for partials result */,
    const half* value, const half* mask, int kv_size, float scale, int reduce_block, int head_stride) {
    const int lane_index = threadIdx.x;
    const int warp_index = threadIdx.y;

    const int warp_data_size = (head_dim*kv_tensor + 2);

    extern __shared__ char shmem[];
    half2* squery2      = (half2*)shmem; // load query buffer
    half * squery       = (half *)shmem; // probabilities buffer after online softmax
    float* sscores      = (float*)(shmem + head_dim*kv_tensor*sizeof(half)); // scores buffer after QK^T
    float* warp_buffer  = (float*)(shmem + head_dim*kv_tensor*sizeof(half) + (kv_block + 2)*sizeof(float) + (warp_index*warp_data_size*sizeof(float)));
#ifndef FA_KV_BLOCK_256
    half*  warp_buffer_half = (half*)warp_buffer;
#endif
    const int HD2 = head_dim / 2;

    // load query with 128x2 shape (repeat row twice)
    const half2* query_ = (const half2*)(query + head_dim*blockIdx.y); // shift as head
#pragma unroll
    for (int j = 0; j < kv_tensor; j += num_warps) {
        const int q_off = j + warp_index;
        if (q_off >= kv_tensor) {
            break;
        }

#pragma unroll
        for (int i = 0; i < HD2; i += WARP_SIZE) {
            const int h_offset = i + lane_index;
            if (h_offset >= HD2) {
                break;
            }
            squery2[q_off*HD2 + h_offset] = query_[h_offset];
        }
    }

    __syncthreads();

    {   // QK^T
        MatrixA query_m;
        nvcuda::wmma::load_matrix_sync(query_m, squery, 16);
        MatrixBT key_m;
        Accum kq_m;

        const int kv_per_warp = kv_block / num_warps;
        const int sum_diag = WMMA_K / kv_tensor;
        // assert(kv_per_warp % kv_tensor == 0);

        const int kvi = warp_index*kv_per_warp;

#pragma unroll
        for (int kv = 0; kv < kv_per_warp; kv += kv_tensor) {
            nvcuda::wmma::load_matrix_sync(key_m, key + head_stride*blockIdx.y + (blockIdx.x*kv_block + kvi + kv)*head_dim, 16);
            nvcuda::wmma::fill_fragment(kq_m, 0.0f);
            nvcuda::wmma::mma_sync(kq_m, query_m, key_m, kq_m);
            nvcuda::wmma::store_matrix_sync(warp_buffer, kq_m, 16, nvcuda::wmma::mem_row_major);

            // sum diagonal
            if (lane_index < kv_tensor) {
                float seq = 0.0f;
                const int seq_idx = kvi + kv + lane_index;
#pragma unroll
                for (int d0 = 0; d0 < sum_diag; d0++) {
                    const int diag_idx = d0 + lane_index * sum_diag;
                    seq += warp_buffer[diag_idx*WMMA_M + diag_idx]; // sum diagonal
                }

                // store sequence result
                sscores[seq_idx] = seq*scale + __half2float(mask[blockIdx.x*kv_block + seq_idx]); // save as float for softmax
            }
        }

        __syncthreads();
    }

    // perform online softmax
    {
        const int kv_per_warp = kv_block / num_warps;
        float M = -INFINITY;

        const int kvi = warp_index*kv_per_warp;

        for (int kv = lane_index*kv_tensor; kv < kv_per_warp; kv += WARP_SIZE*kv_tensor) {
            M = fmaxf(M, fmaxf(sscores[kvi + kv], sscores[kvi + kv + 1]));
        }

        M = warp_reduce_max(M);

        float S = 0.0f;

        for (int kv = lane_index*kv_tensor; kv < kv_per_warp; kv += WARP_SIZE*kv_tensor) {
            S += expf(sscores[kvi + kv] - M);
            S += expf(sscores[kvi + kv + 1] - M);
        }

        S = warp_reduce_sum(S);

        if(lane_index == 0) {
            warp_buffer[0] = M;
            warp_buffer[1] = S;
            // printf("warp index: %d, M= %.4f, S= %.4f\n", warp_index, M, S);
        }

        __syncthreads();

        // reduce warps
        if(warp_index == 0 && lane_index == 0) {
            float M0 = warp_buffer[0];
            float S0 = warp_buffer[1];

            for(int w = 1; w < num_warps; w++) {
                float M1 = warp_buffer[w * warp_data_size];
                float S1 = warp_buffer[w * warp_data_size + 1];

                float M = fmaxf(M0, M1);

                float ms0 = expf(M0 - M);
                float ms1 = expf(M1 - M);

                S0 = S0*ms0 + S1*ms1;
                M0 = M;
            }

            // printf("block M = %.4f, S= %.4f\n", M0, S0);

            // real softmax M and S for this block
            sscores[kv_block] = M0;
            sscores[kv_block + 1] = S0;
        }

        __syncthreads();

        const int tensor_elements = WMMA_M * WMMA_N;

        /*
            [S0, S1, S2,
            S0, S1, S2,
            S0, S1, S2]
        */

        // reuse shared memory padding
        M = sscores[kv_block];
        S = sscores[kv_block + 1];

        const int te_per_warp = tensor_elements / num_warps;

        const int si = warp_index*te_per_warp;

#pragma unroll
        for (int t0 = 0; t0 < te_per_warp; t0 += WARP_SIZE) {
            const int tei = t0 + lane_index;
            if(tei >= te_per_warp) {
                break;
            }

            const int sq_offset = si + tei;
            squery[sq_offset] = __float2half(expf(sscores[sq_offset % kv_block] - M) / S);
        }

        __syncthreads();
    }

#ifdef FA_KV_BLOCK_256
    {  // QK^TV
        MatrixA qk_m;
        nvcuda::wmma::load_matrix_sync(qk_m, squery, 16);
        MatrixBT value_m;
        Accum qkv_m;

        const int reduce_exccedent = reduce_block - gridDim.x;
#pragma unroll
        for(int h0 = 0; h0 < head_dim; h0 += num_warps) {
            const int hi = h0 + warp_index;
            if(hi >= head_dim) {
                break;
            }

            const int output_offset = blockIdx.y * head_stride + hi * reduce_block;

            // `value` need to be transposed
            nvcuda::wmma::load_matrix_sync(value_m, value + hi * kv_size + blockIdx.x*kv_block + blockIdx.y * head_stride, 16);
            nvcuda::wmma::fill_fragment(qkv_m, 0.0f);
            nvcuda::wmma::mma_sync(qkv_m, qk_m, value_m, qkv_m);
            nvcuda::wmma::store_matrix_sync(warp_buffer, qkv_m, 16, nvcuda::wmma::mem_row_major);

            // sum diagonal
            if (lane_index == 0) {
                float hdim = 0.0f;

                for (int d = 0; d < WMMA_K; d++) {
                    hdim += warp_buffer[d*WMMA_M + d]; // sum diagonal
                }

                // assume the key has been processed by blocks launched per head
                key[output_offset + blockIdx.x] = __float2half(hdim);

                if(blockIdx.x == 0) { // just the first block will do this
                    for(int i = 0; i < reduce_exccedent; i ++) {
                        // this is a padding to perform a matrix multiplication without incorrect values
                        key[output_offset + gridDim.x + i] = __float2half(0.0f);
                    }
                }
            }
        }
    }
#else

    { // QK^TV
        MatrixA qk_m;
        nvcuda::wmma::load_matrix_sync(qk_m, squery, 16);
        MatrixB value_m;
        Accum qkv_m;

        const int hd_per_tensor = tensor_elements / kv_block; // head dims processed per tensor core
        const int hd_per_warp = head_dim / num_warps; // head dim split processed per warp
        // assert(hd_per_warp % hd_per_tensor == 0);

        // if(lane_index == 0) {
        //     printf("lidx = %d, widx= %d, hd per tensor= %d, hd per warp = %d\n", lane_index, warp_index, hd_per_tensor, hd_per_warp);
        // }
        const int next_hd_offset = WMMA_N / hd_per_tensor;
        const int sum_diag = WMMA_K / hd_per_tensor;
        const int reduce_exccedent = reduce_block - gridDim.x;

        for (int hdi = warp_index*hd_per_warp; hdi < head_dim; hdi += num_warps*hd_per_warp) {
            // create value matrix in warp buffer KxN
            /*
                head dim (cols)
                [S01, S11, S21, - seq dim (rows)
                S02, S12,  S22,
                S03, S13,  S23]
            */
            for (int hdw = 0; hdw < hd_per_warp; hdw += hd_per_tensor) {
                const int output_offset = blockIdx.y * head_stride + (hdi + hdw + lane_index) * reduce_block;
                if(lane_index < WMMA_N) {
                    for(int r = 0; r < WMMA_K; r++) { // tensor matrix rows
                        // if(lane_index == 15 && r == 15) {
                        //     printf("vidx= %d, vlen= %d, hd_per=%d\n", ((kv_block_offset + r + (c % next_hd_offset)*WMMA_K) * head_dim + (hdi + hdw + c/next_hd_offset)), kv_size*head_dim, hdw);
                        // }
                        warp_buffer_half[r*WMMA_N + lane_index] = value[
                            head_stride*blockIdx.y + // shift head
                            (blockIdx.x*kv_block + r + (lane_index % next_hd_offset)*WMMA_K) * head_dim + // shift sequence
                            (hdi + hdw + lane_index/next_hd_offset)];
                    }
                }

                // perform QK^TV
                nvcuda::wmma::load_matrix_sync(value_m, warp_buffer_half, 16);
                nvcuda::wmma::fill_fragment(qkv_m, 0.0f);
                nvcuda::wmma::mma_sync(qkv_m, qk_m, value_m, qkv_m);
                nvcuda::wmma::store_matrix_sync(warp_buffer, qkv_m, 16, nvcuda::wmma::mem_row_major);

                // sum diagonal
                if (lane_index < hd_per_tensor) {
                    float hdim = 0.0f;
#pragma unroll
                    for (int d0 = 0; d0 < sum_diag; d0++) {
                        const int diag_idx = d0 + lane_index * sum_diag;
                        hdim += warp_buffer[diag_idx*WMMA_M + diag_idx]; // sum diagonal
                    }

                    // assume the key has been processed by blocks launched per head
                    key[output_offset + blockIdx.x] = __float2half(hdim);

                    if(blockIdx.x == 0) { // just the first block will do this
                        for(int i = 0; i < reduce_exccedent; i ++) {
                            // this is a padding to perform a matrix multiplication without incorrect values
                            key[output_offset + gridDim.x + i] = __float2half(0.0f);
                        }
                    }
                }
            }
        }
    }
#endif
}

template<int head_dim, int num_warps>
__global__ void fa_reduce(const half* red_buf, float* qkv, int kv_size, int kv_blocks, int reduce_block) {
    const int lane_index = threadIdx.x;
    const int warp_index = threadIdx.y;

    const int tensor_elements = WMMA_M*WMMA_N;
    const int hi_per_tensor = tensor_elements / reduce_block;

    extern __shared__ char shmem[];
    half * sscale = (half *)shmem;
    float* warp_buffer  = (float*)(shmem + tensor_elements*sizeof(half) + warp_index*tensor_elements*sizeof(float));

    // make scale 1.0 diagonal
    for(int c = warp_index; c < WMMA_K; c += num_warps) {
        if(lane_index < WMMA_M) {
            sscale[c*WMMA_M + lane_index] = __float2half(1.0f);
        }
    }

    __syncthreads();

    MatrixA scale;
    MatrixBT partials;
    nvcuda::wmma::load_matrix_sync(scale, sscale, 16);
    Accum qkv_m;

    const int head_offset = head_dim * kv_size * blockIdx.x;
    const int sum_diag = WMMA_K / hi_per_tensor;

#pragma unroll
    for (int h0 = 0; h0 < head_dim; h0 += num_warps*hi_per_tensor) {
        const int hi = h0 + warp_index*hi_per_tensor;
        nvcuda::wmma::load_matrix_sync(partials, red_buf + (head_offset + hi * reduce_block), 16);
        nvcuda::wmma::fill_fragment(qkv_m, 0.0f);
        nvcuda::wmma::mma_sync(qkv_m, scale, partials, qkv_m);
        nvcuda::wmma::store_matrix_sync(warp_buffer, qkv_m, 16, nvcuda::wmma::mem_row_major);

        // sum diagonal
        if (lane_index < hi_per_tensor) {
            float hdim = 0.0f;

            for (int d = 0; d < sum_diag; d++) {
                const int diag_idx = lane_index * sum_diag + d;
                hdim += warp_buffer[diag_idx*WMMA_M + diag_idx]; // sum diagonal
            }

            qkv[blockIdx.x * head_dim + hi + lane_index] = hdim;
        }
    }
}

int main() {
    print_cuda_info();
    int head_dim = 128, kv_size = 256, num_heads = 1;
    float scale = 1.0f / sqrtf((float)head_dim);
    // allocate memory

    // input buffers
    float* query =  (float*)malloc(head_dim           * num_heads * sizeof(float)); // assume batch size 1
    float* key =    (float*)malloc(head_dim * kv_size * num_heads * sizeof(float));
    float* value =  (float*)malloc(head_dim * kv_size * num_heads * sizeof(float));
    float* mask =   (float*)malloc(kv_size * sizeof(float));

    // output buffers
    float* qkv =    (float*)malloc(head_dim           * num_heads * sizeof(float)); // assume batch size 1
    float* qkv_cuda = (float*)malloc(head_dim           * num_heads * sizeof(float)); // assume batch size 1
    float* scores = (float*)malloc(           kv_size * num_heads * sizeof(float)); // QK^T

    // fill buffers
    fill_buffer(qkv, 0.0f, head_dim * num_heads);
    fill_buffer(scores, 0.0f, kv_size * num_heads);

    random(query, head_dim * num_heads);
    random(key,   head_dim * kv_size * num_heads);
    random(value, head_dim * kv_size * num_heads);
    random(mask,  kv_size);

    if(true) {
        // cpu cmputation
        for(int h = 0; h < num_heads; h++) {
            mulmat_cpu(query + h*head_dim, key + (h * head_dim*kv_size), mask, scores + h*kv_size, 1, kv_size, head_dim, scale, true);
            softmax(scores + h*kv_size, kv_size);
        }

        // print_array("Scores", scores, kv_size, kv_size);

        for(int h = 0; h < num_heads; h++) {
            mulmat_cpu(scores + h*kv_size, value + (h * head_dim*kv_size), nullptr, qkv + h*head_dim, 1, head_dim, kv_size, 1.0f);
        }

        // print_array("QKV", qkv, 8, 16, head_dim);

        fill_buffer(qkv_cuda, 0.0f, head_dim * num_heads);
    }

    if(true) {
        // cuda cumputation
        half * query_f16 =   (half*)malloc(head_dim           * num_heads * sizeof(half));
        half * key_f16 =     (half*)malloc(head_dim * kv_size * num_heads * sizeof(half));
        half * value_f16 =   (half*)malloc(head_dim * kv_size * num_heads * sizeof(half));
        half * mask_f16 =    (half*)malloc(kv_size * sizeof(half));

        for(int i = 0; i < (head_dim           * num_heads); i ++) {
            query_f16[i] = __float2half(query[i]);
        }

        for(int i = 0; i < kv_size; i ++) {
            mask_f16[i] = __float2half(mask[i]);
        }

        for(int i = 0; i < head_dim * kv_size * num_heads; i ++) {
            key_f16[i] = __float2half(key[i]);
#ifndef FA_KV_BLOCK_256
            value_f16[i] = __float2half(value[i]);
#endif
        }

#ifdef FA_KV_BLOCK_256
        // transpose value
        for(int h = 0; h < num_heads; h++) {
            for(int c = 0; c < head_dim; c++) {
                for(int r = 0; r < kv_size; r++) {
                    value_f16[h*kv_size*head_dim + c*kv_size + r] = __float2half(value[h*kv_size*head_dim + r*head_dim + c]);
                }
            }
        }
#endif

        hipStream_t stream;
        hipStreamCreate(&stream);

        half *d_query, *d_key, *d_value, *d_mask;
        float *d_score, *d_qkv;

        hipMalloc((void **)&d_query,   head_dim           * num_heads * sizeof(half));
        hipMalloc((void **)&d_key,     head_dim * kv_size * num_heads * sizeof(half));
        hipMalloc((void **)&d_value,   head_dim * kv_size * num_heads * sizeof(half));
        hipMalloc((void **)&d_mask,    kv_size * sizeof(half));

        hipMalloc((void **)&d_score,              kv_size * num_heads * sizeof(float));
        hipMalloc((void **)&d_qkv,     head_dim           * num_heads * sizeof(float));

        // copy CPU data to GPU memory blocks
        hipMemcpyAsync(d_query, query_f16, head_dim           * num_heads * sizeof(half), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_key,   key_f16,   head_dim * kv_size * num_heads * sizeof(half), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_value, value_f16, head_dim * kv_size * num_heads * sizeof(half), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_mask,  mask_f16,  kv_size * sizeof(half), hipMemcpyHostToDevice, stream);

        constexpr int kv_per_block = 256;
        constexpr int num_warps = 4;

        // assert(kv_size % kv_per_block == 0);
        dim3 grid_dim(kv_size / kv_per_block, num_heads, 1);
        dim3 block_dim(WARP_SIZE, num_warps, 1);

        int shmem =
            head_dim*2*sizeof(half) /* query buffer */ +
            (kv_per_block + 2)*sizeof(float) /* scores buffer */ +
            num_warps * (256 + 2) * sizeof(float) /* tensor core result buffer per warp */;
        printf("\n\nShared memory: %.2f KB\n\n", shmem/1024.0f);

        // print_array("CUDA key", key_f16, 4, 4, head_dim);

        for(int i = 0; i < head_dim * kv_size * num_heads; i ++) {
            key_f16[i] = __float2half(0.0f);
        }

        int reduce_block = ((grid_dim.x + WMMA_M - 1) / WMMA_M) * WMMA_N;
        printf("reduce block: %d\n", reduce_block);

        // half* d_red_buffer;
        // hipMalloc((void **)&d_red_buffer, head_dim * reduce_block * num_heads * sizeof(half));

        hipStreamSynchronize(stream);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, stream);
        flash_attn<128, num_warps, 2, kv_per_block><<<grid_dim, block_dim, shmem, stream>>>
            (d_query, d_key, d_value, d_mask, kv_size, scale, reduce_block, head_dim*kv_size);

        fa_reduce<128, num_warps><<<num_heads, block_dim, shmem, stream>>>(d_key, d_qkv, kv_size, kv_size / kv_per_block, reduce_block);

        hipEventRecord(stop, stream);
        hipEventSynchronize(stop);

        float millis = 0.0f;
        hipEventElapsedTime(&millis, start, stop);

        printf("cuda time: %.4f ms\n", millis);

        // transfer data from device to host
        hipMemcpyAsync(qkv_cuda, d_qkv, head_dim           * num_heads * sizeof(float), hipMemcpyDeviceToHost, stream);

        // half* red_buffer = (half*)malloc(head_dim * reduce_block * num_heads * sizeof(half));
        // hipMemcpyAsync(red_buffer, d_red_buffer, head_dim * reduce_block * num_heads * sizeof(half), hipMemcpyDeviceToHost, stream);

        // hipMemcpyAsync(scores, d_score, kv_size * num_heads * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(key_f16, d_key, head_dim * kv_size * num_heads * sizeof(half), hipMemcpyDeviceToHost, stream);

        hipStreamSynchronize(stream);

        float max_diff = 0.0f;
        int head_idx = 0, dim_idx = 0;

        for(int h = 0; h < num_heads; h++) {
            for(int i = 0; i < head_dim; i++) {
                if(fabs(qkv[h*head_dim + i] - qkv_cuda[h*head_dim + i]) > max_diff) {
                    max_diff = fabs(qkv[h*head_dim + i] - qkv_cuda[h*head_dim + i]);
                    head_idx = h;
                    dim_idx = i;
                }
            }
        }

        printf("R (%.4f) CUDA(%.4f) diff: %.4f - head = %d, dim = %d\n", qkv[head_idx*head_dim + dim_idx], qkv_cuda[head_idx*head_dim + dim_idx], max_diff, head_idx, dim_idx);

        float res = 0.0f;
        for(int i = 0;i < reduce_block; i ++) {
            printf(i < grid_dim.x ? "%.4f " : "[%.4f] ", __half2float(key_f16[head_idx*head_dim*kv_size + dim_idx*reduce_block + i]));
            res += __half2float(key_f16[head_idx*head_dim*kv_size + dim_idx*reduce_block + i]);
        }
        printf(" = %.4f\n", res);
        // print_array("CUDA QKV", qkv_cuda, 8, 16, head_dim);

        // clean up device memory
        hipFree(d_query);
        hipFree(d_key);
        hipFree(d_value);
        hipFree(d_qkv);
        hipFree(d_score);
    }

    free(query);
    free(key);
    free(value);
    free(qkv);
    free(qkv_cuda);
    free(scores);
    return 0;
}