#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>
#include "cuda_info.h"
#include "utils.h"

#define WARP_SIZE 32
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

typedef nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, nvcuda::wmma::row_major> MatrixA;
typedef nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, nvcuda::wmma::col_major> MatrixBT;
typedef nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, nvcuda::wmma::row_major> MatrixB;
typedef nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float>                          Accum;

// static __device__ __forceinline__ float warp_reduce_sum(float x) {
// #pragma unroll
//     for (int mask = 16; mask > 0; mask >>= 1) {
//         x += __shfl_xor_sync(0xffffffff, x, mask, 32);
//     }
//     return x;
// }

#define FA_KV_BLOCK_256

template<int head_dim, int num_warps, int kv_tensor, int kv_block>
__global__ void flash_attn(const half* query,
    half* key /* reuse key buffer for partials result */,
    const half* value, int kv_size, int reduce_block, int head_stride) {
    const int lane_index = threadIdx.x;
    const int warp_index = threadIdx.y;

    extern __shared__ char shmem[];
    half2* squery2      = (half2*)shmem; // load query buffer
    half * squery       = (half *)shmem; // probabilities buffer after online softmax
    float* sscores      = (float*)(shmem + head_dim*kv_tensor*sizeof(half)); // scores buffer after QK^T
    float* warp_buffer  = (float*)(shmem + head_dim*kv_tensor*sizeof(half) + kv_block*sizeof(float) + (warp_index*head_dim*kv_tensor*sizeof(float)));
#ifndef FA_KV_BLOCK_256
    half*  warp_buffer_half = (half*)warp_buffer;
#endif
    const int HD2 = head_dim / 2;

    // load query with 128x2 shape (repeat row twice)
    const half2* query_ = (const half2*)(query + head_dim*blockIdx.y); // shift as head
#pragma unroll
    for (int j = 0; j < kv_tensor; j += num_warps) {
        const int q_off = j + warp_index;
        if (q_off >= kv_tensor) {
            break;
        }

#pragma unroll
        for (int i = 0; i < HD2; i += WARP_SIZE) {
            const int h_offset = i + lane_index;
            if (h_offset >= HD2) {
                break;
            }
            squery2[q_off*HD2 + h_offset] = query_[h_offset];
        }
    }

    __syncthreads();

    {   // QK^T
        MatrixA query_m;
        nvcuda::wmma::load_matrix_sync(query_m, squery, 16);
        MatrixBT key_m;
        Accum kq_m;

        const int kv_per_warp = kv_block / num_warps;
        const int sum_diag = WMMA_K / kv_tensor;
        // assert(kv_per_warp % kv_tensor == 0);

        for (int kvi = warp_index*kv_per_warp; kvi < kv_block; kvi += num_warps*kv_per_warp) {
            for (int kv = 0; kv < kv_per_warp; kv += kv_tensor) {
                nvcuda::wmma::load_matrix_sync(key_m, key + head_stride*blockIdx.y + (blockIdx.x*kv_block + kvi + kv)*head_dim, 16);
                nvcuda::wmma::fill_fragment(kq_m, 0.0f);
                nvcuda::wmma::mma_sync(kq_m, query_m, key_m, kq_m);
                nvcuda::wmma::store_matrix_sync(warp_buffer, kq_m, 16, nvcuda::wmma::mem_row_major);

                // sum diagonal
                if (lane_index < kv_tensor) {
                    float seq = 0.0f;
    #pragma unroll
                    for (int d0 = 0; d0 < sum_diag; d0++) {
                        const int diag_idx = d0 + lane_index * sum_diag;
                        seq += warp_buffer[diag_idx*WMMA_M + diag_idx]; // sum diagonal
                    }
                    sscores[kvi + kv + lane_index] = seq; // save as float for softmax
                }
            }
        }

        __syncthreads();
    }

    const int tensor_elements = WMMA_M*WMMA_N;

    { // fill `squery` buffer with scores (repeat if is needed)
        /*
            [S0, S1, S2,
            S0, S1, S2,
            S0, S1, S2]
        */
        const int te_per_warp = tensor_elements / num_warps;
        for (int si = warp_index*te_per_warp; si < tensor_elements; si += num_warps*te_per_warp) {
            for (int tei = lane_index; tei < te_per_warp; tei += WARP_SIZE) {
                const int sq_offset = si + tei;
                squery[sq_offset] = __float2half(sscores[sq_offset % kv_block]);
            }
        }
    }

#ifdef FA_KV_BLOCK_256
    {  // QK^TV
        MatrixA qk_m;
        nvcuda::wmma::load_matrix_sync(qk_m, squery, 16);
        MatrixBT value_m;
        Accum qkv_m;

        const int reduce_exccedent = reduce_block - gridDim.x;

        for(int hi = warp_index; hi < head_dim; hi += num_warps) {
            const int output_offset = blockIdx.y * head_dim * kv_size + hi * reduce_block;

            // `value` need to be transposed
            nvcuda::wmma::load_matrix_sync(value_m, value + hi * kv_size + blockIdx.x*kv_block + blockIdx.y * head_stride, 16);
            nvcuda::wmma::fill_fragment(qkv_m, 0.0f);
            nvcuda::wmma::mma_sync(qkv_m, qk_m, value_m, qkv_m);
            nvcuda::wmma::store_matrix_sync(warp_buffer, qkv_m, 16, nvcuda::wmma::mem_row_major);

            // sum diagonal
            if (lane_index == 0) {
                float hdim = 0.0f;

                for (int d = 0; d < WMMA_K; d++) {
                    hdim += warp_buffer[d*WMMA_M + d]; // sum diagonal
                }

                // assume the key has been processed by blocks launched per head
                key[output_offset + blockIdx.x] = __float2half(hdim);

                if(blockIdx.x == 0) { // just the first block will do this
                    for(int i = 0; i < reduce_exccedent; i ++) {
                        // this is a padding to perform a matrix multiplication without incorrect values
                        key[output_offset + gridDim.x + i] = __float2half(0.0f);
                    }
                }
            }
        }
    }
#else

    { // QK^TV
        MatrixA qk_m;
        nvcuda::wmma::load_matrix_sync(qk_m, squery, 16);
        MatrixB value_m;
        Accum qkv_m;

        const int hd_per_tensor = tensor_elements / kv_block; // head dims processed per tensor core
        const int hd_per_warp = head_dim / num_warps; // head dim split processed per warp
        // assert(hd_per_warp % hd_per_tensor == 0);

        // if(lane_index == 0) {
        //     printf("lidx = %d, widx= %d, hd per tensor= %d, hd per warp = %d\n", lane_index, warp_index, hd_per_tensor, hd_per_warp);
        // }
        const int next_hd_offset = WMMA_N / hd_per_tensor;
        const int sum_diag = WMMA_K / hd_per_tensor;
        const int reduce_exccedent = reduce_block - gridDim.x;

        for (int hdi = warp_index*hd_per_warp; hdi < head_dim; hdi += num_warps*hd_per_warp) {
            // create value matrix in warp buffer KxN
            /*
                head dim (cols)
                [S01, S11, S21, - seq dim (rows)
                S02, S12,  S22,
                S03, S13,  S23]
            */
            for (int hdw = 0; hdw < hd_per_warp; hdw += hd_per_tensor) {
                const int output_offset = blockIdx.y * head_stride + (hdi + hdw + lane_index) * reduce_block;
                if(lane_index < WMMA_N) {
                    for(int r = 0; r < WMMA_K; r++) { // tensor matrix rows
                        // if(lane_index == 15 && r == 15) {
                        //     printf("vidx= %d, vlen= %d, hd_per=%d\n", ((kv_block_offset + r + (c % next_hd_offset)*WMMA_K) * head_dim + (hdi + hdw + c/next_hd_offset)), kv_size*head_dim, hdw);
                        // }
                        warp_buffer_half[r*WMMA_N + lane_index] = value[
                            head_stride*blockIdx.y + // shift head
                            (blockIdx.x*kv_block + r + (lane_index % next_hd_offset)*WMMA_K) * head_dim + // shift sequence
                            (hdi + hdw + lane_index/next_hd_offset)];
                    }
                }

                // perform QK^TV
                nvcuda::wmma::load_matrix_sync(value_m, warp_buffer_half, 16);
                nvcuda::wmma::fill_fragment(qkv_m, 0.0f);
                nvcuda::wmma::mma_sync(qkv_m, qk_m, value_m, qkv_m);
                nvcuda::wmma::store_matrix_sync(warp_buffer, qkv_m, 16, nvcuda::wmma::mem_row_major);

                // sum diagonal
                if (lane_index < hd_per_tensor) {
                    float hdim = 0.0f;
#pragma unroll
                    for (int d0 = 0; d0 < sum_diag; d0++) {
                        const int diag_idx = d0 + lane_index * sum_diag;
                        hdim += warp_buffer[diag_idx*WMMA_M + diag_idx]; // sum diagonal
                    }

                    // assume the key has been processed by blocks launched per head
                    key[output_offset + blockIdx.x] = __float2half(hdim);

                    if(blockIdx.x == 0) { // just the first block will do this
                        for(int i = 0; i < reduce_exccedent; i ++) {
                            // this is a padding to perform a matrix multiplication without incorrect values
                            key[output_offset + gridDim.x + i] = __float2half(0.0f);
                        }
                    }
                }
            }
        }
    }
#endif
}

template<int head_dim, int num_warps>
__global__ void fa_reduce(const half* key, float* qkv, int kv_size, int kv_blocks, int reduce_block) {
    const int lane_index = threadIdx.x;
    const int warp_index = threadIdx.y;

    const int tensor_elements = WMMA_M*WMMA_N;
    const int hi_per_tensor = tensor_elements / reduce_block;

    extern __shared__ char shmem[];
    half * sscale = (half *)shmem;
    float* warp_buffer  = (float*)(shmem + tensor_elements*sizeof(half) + warp_index*tensor_elements*sizeof(float));

    // make scale 1.0 diagonal
    for(int c = warp_index; c < WMMA_K; c += num_warps) {
        if(lane_index < WMMA_M) {
            sscale[c*WMMA_M + lane_index] = __float2half(1.0f);
        }
    }

    __syncthreads();

    MatrixA scale;
    MatrixBT partials;
    nvcuda::wmma::load_matrix_sync(scale, sscale, 16);
    Accum qkv_m;

    const int head_offset = head_dim * kv_size * blockIdx.x;
    const int sum_diag = WMMA_K / hi_per_tensor;

    for (int hi = warp_index*hi_per_tensor; hi < head_dim; hi += num_warps*hi_per_tensor) {
        nvcuda::wmma::load_matrix_sync(partials, key + (head_offset + hi * reduce_block), 16);
        nvcuda::wmma::fill_fragment(qkv_m, 0.0f);
        nvcuda::wmma::mma_sync(qkv_m, scale, partials, qkv_m);
        nvcuda::wmma::store_matrix_sync(warp_buffer, qkv_m, 16, nvcuda::wmma::mem_row_major);

        // sum diagonal
        if (lane_index < hi_per_tensor) {
            float hdim = 0.0f;

            for (int d = 0; d < sum_diag; d++) {
                const int diag_idx = lane_index * sum_diag + d;
                hdim += warp_buffer[diag_idx*WMMA_M + diag_idx]; // sum diagonal
            }

            qkv[blockIdx.x * head_dim + hi + lane_index] = hdim;
        }
    }
}

int main() {
    print_cuda_info();
    int head_dim = 128, kv_size = 4096, num_heads = 32;
    // allocate memory

    // input buffers
    float* query =  (float*)malloc(head_dim           * num_heads * sizeof(float)); // assume batch size 1
    float* key =    (float*)malloc(head_dim * kv_size * num_heads * sizeof(float));
    float* value =  (float*)malloc(head_dim * kv_size * num_heads * sizeof(float));

    // output buffers
    float* qkv =    (float*)malloc(head_dim           * num_heads * sizeof(float)); // assume batch size 1
    float* scores = (float*)malloc(           kv_size * num_heads * sizeof(float)); // QK^T

    // fill buffers
    fill_buffer(qkv, 0.0f, head_dim * num_heads);
    fill_buffer(scores, 0.0f, kv_size * num_heads);

    random(query, head_dim * num_heads);
    random(key,   head_dim * kv_size * num_heads);
    random(value, head_dim * kv_size * num_heads);

    if(true) {
        // cpu cmputation
        for(int h = 0; h < num_heads; h++) {
            mulmat_cpu(query + h*head_dim, key + (h * head_dim*kv_size), scores + h*kv_size, 1, kv_size, head_dim, true);
        }

        // print_array("Scores", scores, kv_size, kv_size);

        for(int h = 0; h < num_heads; h++) {
            mulmat_cpu(scores + h*kv_size, value + (h * head_dim*kv_size), qkv + h*head_dim, 1, head_dim, kv_size);
        }

        print_array("QKV", qkv, 8, 16, head_dim);

        fill_buffer(qkv, 0.0f, head_dim * num_heads);
    }

    if(true) {
        // cuda cumputation
        half * query_f16 =   (half*)malloc(head_dim           * num_heads * sizeof(half));
        half * key_f16 =     (half*)malloc(head_dim * kv_size * num_heads * sizeof(half));
        half * value_f16 =   (half*)malloc(head_dim * kv_size * num_heads * sizeof(half));

        for(int i = 0; i < (head_dim           * num_heads); i ++) {
            query_f16[i] = __float2half(query[i]);
        }

        for(int i = 0; i < head_dim * kv_size * num_heads; i ++) {
            key_f16[i] = __float2half(key[i]);
#ifndef FA_KV_BLOCK_256
            value_f16[i] = __float2half(value[i]);
#endif
        }
#ifdef FA_KV_BLOCK_256
        // transpose value
        for(int h = 0; h < num_heads; h++) {
            for(int c = 0; c < head_dim; c++) {
                for(int r = 0; r < kv_size; r++) {
                    value_f16[h*kv_size*head_dim + c*kv_size + r] = __float2half(value[h*kv_size*head_dim + r*head_dim + c]);
                }
            }
        }
#endif

        hipStream_t stream;
        hipStreamCreate(&stream);

        half *d_query, *d_key, *d_value;
        float *d_score, *d_qkv;

        hipMalloc((void **)&d_query,   head_dim           * num_heads * sizeof(half));
        hipMalloc((void **)&d_key,     head_dim * kv_size * num_heads * sizeof(half));
        hipMalloc((void **)&d_value,   head_dim * kv_size * num_heads * sizeof(half));

        hipMalloc((void **)&d_score,              kv_size * num_heads * sizeof(float));
        hipMalloc((void **)&d_qkv,     head_dim           * num_heads * sizeof(float));

        // copy CPU data to GPU memory blocks
        hipMemcpyAsync(d_query, query_f16, head_dim           * num_heads * sizeof(half), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_key,   key_f16,   head_dim * kv_size * num_heads * sizeof(half), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_value, value_f16, head_dim * kv_size * num_heads * sizeof(half), hipMemcpyHostToDevice, stream);

        constexpr int kv_per_block = 256;
        constexpr int num_warps = 4;

        // assert(kv_size % kv_per_block == 0)
        dim3 grid_dim(kv_size / kv_per_block, num_heads, 1);
        dim3 block_dim(WARP_SIZE, num_warps, 1);

        int shmem =
            head_dim*2*sizeof(half) /* query buffer */ +
            kv_per_block*sizeof(float) /* scores buffer */ +
            num_warps*256*sizeof(float) /* tensor core result buffer per warp */;
        printf("\n\nShared memory: %.2f KB\n\n", shmem/1024.0f);

        // print_array("CUDA key", key_f16, 4, 4, head_dim);

        for(int i = 0; i < head_dim * kv_size * num_heads; i ++) {
            key_f16[i] = __float2half(0.0f);
        }

        int reduce_block = ((grid_dim.x + WMMA_M - 1) / WMMA_M) * WMMA_N;
        printf("reduce block: %d\n", reduce_block);

        flash_attn<128, num_warps, 2, kv_per_block><<<grid_dim, block_dim, shmem, stream>>>(d_query, d_key, d_value, kv_size, reduce_block, head_dim*kv_size);

        fa_reduce<128, num_warps><<<num_heads, block_dim, shmem, stream>>>(d_key, d_qkv, kv_size, kv_size / kv_per_block, reduce_block);

        // transfer data from device to host
        hipMemcpyAsync(qkv, d_qkv, head_dim           * num_heads * sizeof(float), hipMemcpyDeviceToHost, stream);
        // hipMemcpyAsync(scores, d_score, kv_size * num_heads * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(key_f16, d_key, head_dim * kv_size * num_heads * sizeof(half), hipMemcpyDeviceToHost, stream);

        hipStreamSynchronize(stream);

        print_array("CUDA QKV", qkv, 8, 16, head_dim);

        // clean up device memory
        hipFree(d_query);
        hipFree(d_key);
        hipFree(d_value);
        hipFree(d_qkv);
        hipFree(d_score);
    }

    free(query);
    free(key);
    free(value);
    free(qkv);
    free(scores);
    return 0;
}