#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>
#include "cuda_info.h"
#include "utils.h"
#include "tensor-mma.h"
#include "flash-llama.h"

#define FA_KV_BLOCK_256

template<int head_dim, int num_warps, int kv_tensor, int kv_block>
__global__ void flash_attn(half* query,
    half* key /* reuse key buffer for partials result */,
    const half* value, const half* mask, int kv_size, float scale, int reduce_block, int head_stride) {
    const int lane_index = threadIdx.x;
    const int warp_index = threadIdx.y;

    const int warp_data_size = (head_dim*kv_tensor + 2);

    extern __shared__ char shmem[];
    half2* squery2      = (half2*)shmem; // load query buffer
    half * squery       = (half *)shmem; // probabilities buffer after online softmax
    float* sscores      = (float*)(shmem + head_dim*kv_tensor*sizeof(half)); // scores buffer after QK^T
    float* warp_buffer  = (float*)(shmem + head_dim*kv_tensor*sizeof(half) + (kv_block + 2)*sizeof(float) + (warp_index*warp_data_size*sizeof(float)));
#ifndef FA_KV_BLOCK_256
    half*  warp_buffer_half = (half*)warp_buffer;
#endif
    const int HD2 = head_dim / 2;

    // load query with 128x2 shape (repeat row twice)
    const half2* query_ = (const half2*)(query + head_dim*blockIdx.y); // shift as head
#pragma unroll
    for (int j = 0; j < kv_tensor; j += num_warps) {
        const int q_off = j + warp_index;
        if (q_off >= kv_tensor) {
            break;
        }

#pragma unroll
        for (int i = 0; i < HD2; i += WARP_SIZE) {
            const int h_offset = i + lane_index;
            if (h_offset >= HD2) {
                break;
            }
            squery2[q_off*HD2 + h_offset] = query_[h_offset];
        }
    }

    __syncthreads();

    {   // QK^T
        MatrixA query_m;
        nvcuda::wmma::load_matrix_sync(query_m, squery, 16);
        MatrixBT key_m;
        Accum kq_m;

        const int kv_per_warp = kv_block / num_warps;
        const int sum_diag = WMMA_K / kv_tensor;
        // assert(kv_per_warp % kv_tensor == 0);

        const int kvi = warp_index*kv_per_warp;

#pragma unroll
        for (int kv = 0; kv < kv_per_warp; kv += kv_tensor) {
            nvcuda::wmma::load_matrix_sync(key_m, key + head_stride*blockIdx.y + (blockIdx.x*kv_block + kvi + kv)*head_dim, 16);
            nvcuda::wmma::fill_fragment(kq_m, 0.0f);
            nvcuda::wmma::mma_sync(kq_m, query_m, key_m, kq_m);
            nvcuda::wmma::store_matrix_sync(warp_buffer, kq_m, 16, nvcuda::wmma::mem_row_major);

            // sum diagonal
            if (lane_index < kv_tensor) {
                float seq = 0.0f;
                const int seq_idx = kvi + kv + lane_index;
#pragma unroll
                for (int d0 = 0; d0 < sum_diag; d0++) {
                    const int diag_idx = d0 + lane_index * sum_diag;
                    seq += warp_buffer[diag_idx*WMMA_M + diag_idx]; // sum diagonal
                }

                // store sequence result
                sscores[seq_idx] = seq*scale + __half2float(mask[blockIdx.x*kv_block + seq_idx]); // save as float for softmax
            }
        }

        __syncthreads();
    }

    // perform online softmax
    {
        const int kv_per_warp = kv_block / num_warps;
        float M = -INFINITY;

        const int kvi = warp_index*kv_per_warp;

        for (int kv = lane_index*kv_tensor; kv < kv_per_warp; kv += WARP_SIZE*kv_tensor) {
            M = fmaxf(M, fmaxf(sscores[kvi + kv], sscores[kvi + kv + 1]));
        }

        M = warp_reduce_max(M);

        float S = 0.0f;

        for (int kv = lane_index*kv_tensor; kv < kv_per_warp; kv += WARP_SIZE*kv_tensor) {
            S += expf(sscores[kvi + kv] - M);
            S += expf(sscores[kvi + kv + 1] - M);
        }

        S = warp_reduce_sum(S);

        if(lane_index == 0) {
            warp_buffer[0] = M;
            warp_buffer[1] = S;
            // printf("warp index: %d, M= %.4f, S= %.4f\n", warp_index, M, S);
        }

        __syncthreads();

        // reduce warps
        if(warp_index == 0 && lane_index == 0) {
            float M0 = warp_buffer[0];
            float S0 = warp_buffer[1];

            for(int w = 1; w < num_warps; w++) {
                float M1 = warp_buffer[w * warp_data_size];
                float S1 = warp_buffer[w * warp_data_size + 1];

                float M = fmaxf(M0, M1);

                float ms0 = expf(M0 - M);
                float ms1 = expf(M1 - M);

                S0 = S0*ms0 + S1*ms1;
                M0 = M;
            }

            // printf("block M = %.4f, S= %.4f\n", M0, S0);

            // real softmax M and S for this block
            sscores[kv_block] = M0;
            sscores[kv_block + 1] = S0;
        }

        __syncthreads();

        const int tensor_elements = WMMA_M * WMMA_N;

        /*

            [S0, S1, S2,
            S0, S1, S2,
            S0, S1, S2]

        */

        // reuse shared memory padding
        M = sscores[kv_block];
        // S = sscores[kv_block + 1];

        const int te_per_warp = tensor_elements / num_warps;

        const int si = warp_index*te_per_warp;

#pragma unroll
        for (int t0 = 0; t0 < te_per_warp; t0 += WARP_SIZE) {
            const int tei = t0 + lane_index;
            if(tei >= te_per_warp) {
                break;
            }

            const int sq_offset = si + tei;
            squery[sq_offset] = __float2half(expf(sscores[sq_offset % kv_block] - M));
        }

        __syncthreads();
    }

#ifdef FA_KV_BLOCK_256
    {  // QK^TV
        MatrixA qk_m;
        nvcuda::wmma::load_matrix_sync(qk_m, squery, 16);
        MatrixBT value_m;
        Accum qkv_m;

        const int reduce_exccedent = reduce_block - gridDim.x;
#pragma unroll
        for(int h0 = 0; h0 < head_dim; h0 += num_warps) {
            const int hi = h0 + warp_index;
            if(hi >= head_dim) {
                break;
            }

            const int output_offset = blockIdx.y * head_stride + hi * reduce_block;

            // `value` need to be transposed
            nvcuda::wmma::load_matrix_sync(value_m, value + hi * kv_size + blockIdx.x*kv_block + blockIdx.y * head_stride, 16);
            nvcuda::wmma::fill_fragment(qkv_m, 0.0f);
            nvcuda::wmma::mma_sync(qkv_m, qk_m, value_m, qkv_m);
            nvcuda::wmma::store_matrix_sync(warp_buffer, qkv_m, 16, nvcuda::wmma::mem_row_major);

            // sum diagonal
            if (lane_index == 0) {
                float hdim = 0.0f;

                for (int d = 0; d < WMMA_K; d++) {
                    hdim += warp_buffer[d*WMMA_M + d]; // sum diagonal
                }

                // float hdim2 = 0.0f;
                // for (int d = 0; d < WMMA_K; d++) {
                //     if(d < 8) {
                //         hdim += warp_buffer[d*WMMA_M + d];
                //     } else {
                //         hdim2 += warp_buffer[d*WMMA_M + d];
                //     }
                // }

                // printf("warp 0 dim %d: %.4f\nwarp 1 dim %d: %.4f\n", hi, hdim, hi, hdim2);
                // float real_dim = hdim*__half2float(query[0]) + hdim2*__half2float(query[1]);
                // printf("real dim %d = %.4f, S=%.4f\n", hi, real_dim, sscores[kv_block + 1]);

                // assume the key has been processed by blocks launched per head
                key[output_offset + blockIdx.x] = __float2half(hdim);
                key[blockIdx.y * head_stride + head_dim*reduce_block + blockIdx.x*2] = __float2half(sscores[kv_block]); // max of this kv block
                key[blockIdx.y * head_stride + head_dim*reduce_block + blockIdx.x*2 + 1] = __float2half(sscores[kv_block + 1]); // sum of this kv block

                if(blockIdx.x == 0) { // just the first block will do this
                    for(int i = 0; i < reduce_exccedent; i ++) {
                        // this is a padding to perform a matrix multiplication without incorrect values
                        key[output_offset + gridDim.x + i] = __float2half(0.0f);
                    }
                }
            }
        }
    }
#else

    { // QK^TV
        MatrixA qk_m;
        nvcuda::wmma::load_matrix_sync(qk_m, squery, 16);
        MatrixB value_m;
        Accum qkv_m;

        const int hd_per_tensor = tensor_elements / kv_block; // head dims processed per tensor core
        const int hd_per_warp = head_dim / num_warps; // head dim split processed per warp
        // assert(hd_per_warp % hd_per_tensor == 0);

        // if(lane_index == 0) {
        //     printf("lidx = %d, widx= %d, hd per tensor= %d, hd per warp = %d\n", lane_index, warp_index, hd_per_tensor, hd_per_warp);
        // }
        const int next_hd_offset = WMMA_N / hd_per_tensor;
        const int sum_diag = WMMA_K / hd_per_tensor;
        const int reduce_exccedent = reduce_block - gridDim.x;

        for (int hdi = warp_index*hd_per_warp; hdi < head_dim; hdi += num_warps*hd_per_warp) {
            // create value matrix in warp buffer KxN
            /*
                head dim (cols)
                [S01, S11, S21, - seq dim (rows)
                S02, S12,  S22,
                S03, S13,  S23]
            */
            for (int hdw = 0; hdw < hd_per_warp; hdw += hd_per_tensor) {
                const int output_offset = blockIdx.y * head_stride + (hdi + hdw + lane_index) * reduce_block;
                if(lane_index < WMMA_N) {
                    for(int r = 0; r < WMMA_K; r++) { // tensor matrix rows
                        // if(lane_index == 15 && r == 15) {
                        //     printf("vidx= %d, vlen= %d, hd_per=%d\n", ((kv_block_offset + r + (c % next_hd_offset)*WMMA_K) * head_dim + (hdi + hdw + c/next_hd_offset)), kv_size*head_dim, hdw);
                        // }
                        warp_buffer_half[r*WMMA_N + lane_index] = value[
                            head_stride*blockIdx.y + // shift head
                            (blockIdx.x*kv_block + r + (lane_index % next_hd_offset)*WMMA_K) * head_dim + // shift sequence
                            (hdi + hdw + lane_index/next_hd_offset)];
                    }
                }

                // perform QK^TV
                nvcuda::wmma::load_matrix_sync(value_m, warp_buffer_half, 16);
                nvcuda::wmma::fill_fragment(qkv_m, 0.0f);
                nvcuda::wmma::mma_sync(qkv_m, qk_m, value_m, qkv_m);
                nvcuda::wmma::store_matrix_sync(warp_buffer, qkv_m, 16, nvcuda::wmma::mem_row_major);

                // sum diagonal
                if (lane_index < hd_per_tensor) {
                    float hdim = 0.0f;
#pragma unroll
                    for (int d0 = 0; d0 < sum_diag; d0++) {
                        const int diag_idx = d0 + lane_index * sum_diag;
                        hdim += warp_buffer[diag_idx*WMMA_M + diag_idx]; // sum diagonal
                    }

                    // assume the key has been processed by blocks launched per head
                    key[output_offset + blockIdx.x] = __float2half(hdim);

                    if(blockIdx.x == 0) { // just the first block will do this
                        for(int i = 0; i < reduce_exccedent; i ++) {
                            // this is a padding to perform a matrix multiplication without incorrect values
                            key[output_offset + gridDim.x + i] = __float2half(0.0f);
                        }
                    }
                }
            }
        }
    }
#endif
}

template<int head_dim, int num_warps>
__global__ void fa_reduce(const half* red_buf, float* qkv, int kv_size, int num_kv_blocks, int reduce_block) {
    const int lane_index = threadIdx.x;
    const int warp_index = threadIdx.y;

    const int tensor_elements = WMMA_M*WMMA_N;
    //const int hi_per_tensor = tensor_elements / reduce_block;
    const int head_offset = head_dim * kv_size * blockIdx.x;

    extern __shared__ char shmem[];
    half * sscale = (half *)shmem;
    float* sf_lse = (float*)(shmem + tensor_elements*sizeof(half));
    float* warp_buffer  = (float*)(shmem + tensor_elements*sizeof(half) + (kv_size/reduce_block + 2) * sizeof(float) + warp_index*(tensor_elements + 2)*sizeof(float));

    // make scale 1.0 diagonal
    if(warp_index == 0 && lane_index == 0) {
        const int softmax_lse_offset = head_offset + head_dim*reduce_block;
        float M0 = __half2float(red_buf[softmax_lse_offset]);
        float S0 = __half2float(red_buf[softmax_lse_offset + 1]);

        for(int i = 1; i < num_kv_blocks; i++) {
            float M1 = __half2float(red_buf[softmax_lse_offset + i*2]);
            float S1 = __half2float(red_buf[softmax_lse_offset + i*2 + 1]);

            float M = fmaxf(M0, M1);

            float ms0 = expf(M0 - M);
            float ms1 = expf(M1 - M);

            S0 = S0*ms0 + S1*ms1;
            M0 = M;

            sscale[i*2    ] = __float2half(ms0);
            sscale[i*2 + 1] = __float2half(ms1);
        }

        sf_lse[0] = S0;

        // S0 is all sequence softmax denominator
        // printf("CUDA S: %.4f M: %.4f\n", S0, M0);
    }

    __syncthreads();

    const int hd_per_warp = head_dim / num_warps;

    // reduce kv blocks (very slow!!)
    for(int hi = warp_index*hd_per_warp; hi < head_dim; hi += num_warps*hd_per_warp) {
        for(int hdi = lane_index; hdi < hd_per_warp; hdi += WARP_SIZE) {
            float hdim = __half2float(red_buf[head_offset + (hi + hdi) * reduce_block]);
            for(int kv = 1; kv < num_kv_blocks; kv++) {
                hdim = hdim*__half2float(sscale[kv*2]) + __half2float(red_buf[head_offset + (hi + hdi) * reduce_block + kv]) * __half2float(sscale[kv*2 + 1]);
            }
            qkv[blockIdx.x * head_dim + hi + lane_index] = hdim / sf_lse[0];
        }
    }

    // for(int j = 1 + warp_index; j < hi_per_tensor; j += num_warps) {
    //     for(int i = lane_index; i < reduce_block; i += WARP_SIZE) {
    //         sscale[j * reduce_block + i] = sscale[i];
    //     }
    // }

//     MatrixA scale;
//     MatrixBT partials;
//     nvcuda::wmma::load_matrix_sync(scale, sscale, 16);
//     Accum qkv_m;

//     const int sum_diag = WMMA_K / hi_per_tensor;

// #pragma unroll
//     for (int h0 = 0; h0 < head_dim; h0 += num_warps*hi_per_tensor) {
//         const int hi = h0 + warp_index*hi_per_tensor;
//         nvcuda::wmma::load_matrix_sync(partials, red_buf + (head_offset + hi * reduce_block), 16);
//         nvcuda::wmma::fill_fragment(qkv_m, 0.0f);
//         nvcuda::wmma::mma_sync(qkv_m, scale, partials, qkv_m);
//         nvcuda::wmma::store_matrix_sync(warp_buffer, qkv_m, 16, nvcuda::wmma::mem_row_major);

//         // sum diagonal
//         if (lane_index < hi_per_tensor) {
//             float hdim = 0.0f;

//             for (int d = 0; d < sum_diag; d++) {
//                 const int diag_idx = lane_index * sum_diag + d;
//                 hdim += warp_buffer[diag_idx*WMMA_M + diag_idx]; // sum diagonal
//             }

//             qkv[blockIdx.x * head_dim + hi + lane_index] = hdim / sf_lse[0];
//         }
//     }
}

int main() {
    print_cuda_info();
    int head_dim = 128, kv_size = 2048, num_heads = 32;
    float scale = 1.0f / sqrtf((float)head_dim);
    // allocate memory

    // input buffers
    float* query =  (float*)malloc(head_dim           * num_heads * sizeof(float)); // assume batch size 1
    float* key =    (float*)malloc(head_dim * kv_size * num_heads * sizeof(float));
    float* value =  (float*)malloc(head_dim * kv_size * num_heads * sizeof(float));
    float* mask =   (float*)malloc(kv_size * sizeof(float));

    // output buffers
    float* qkv =    (float*)malloc(head_dim           * num_heads * sizeof(float)); // assume batch size 1
    float* qkv_cuda = (float*)malloc(head_dim           * num_heads * sizeof(float)); // assume batch size 1
    float* scores = (float*)malloc(           kv_size * num_heads * sizeof(float)); // QK^T

    // fill buffers
    fill_buffer(qkv, 0.0f, head_dim * num_heads);
    fill_buffer(scores, 0.0f, kv_size * num_heads);

    random(query, head_dim * num_heads);
    random(key,   head_dim * kv_size * num_heads);
    random(value, head_dim * kv_size * num_heads);
    random(mask,  kv_size);

    if(true) {
        // cpu cmputation
        for(int h = 0; h < num_heads; h++) {
            mulmat_cpu(query + h*head_dim, key + (h * head_dim*kv_size), mask, scores + h*kv_size, 1, kv_size, head_dim, scale, true);
            softmax(scores + h*kv_size, kv_size);
        }

        // print_array("Scores", scores, kv_size, 8);

        for(int h = 0; h < num_heads; h++) {
            mulmat_cpu(scores + h*kv_size, value + (h * head_dim*kv_size), nullptr, qkv + h*head_dim, 1, head_dim, kv_size, 1.0f);
        }

        print_array("QKV", qkv, 2, 16, head_dim);

        fill_buffer(qkv_cuda, 0.0f, head_dim * num_heads);
    }

    if(true) {
        // cuda cumputation
        half * query_f16 =   (half*)malloc(head_dim           * num_heads * sizeof(half));
        half * key_f16 =     (half*)malloc(head_dim * kv_size * num_heads * sizeof(half));
        half * value_f16 =   (half*)malloc(head_dim * kv_size * num_heads * sizeof(half));
        half * value_f16_nT =   (half*)malloc(head_dim * kv_size * num_heads * sizeof(half));
        half * mask_f16 =    (half*)malloc(kv_size * sizeof(half));
        half * mask_f16_padded = (half*)malloc(kv_size * 16 * sizeof(half));

        for(int i = 0; i < (head_dim           * num_heads); i ++) {
            query_f16[i] = __float2half(query[i]);
        }

        for(int b = 0; b < 16; b ++) {
            for(int i = 0; i < kv_size; i ++) {
                if(b == 0) {
                    mask_f16[i] = __float2half(mask[i]);
                    mask_f16_padded[i] = __float2half(mask[i]);
                } else {
                    mask_f16_padded[b*kv_size + i] =  __float2half(0.0f);
                }
            }
        }

        for(int i = 0; i < head_dim * kv_size * num_heads; i ++) {
            key_f16[i] = __float2half(key[i]);
#ifndef FA_KV_BLOCK_256
            value_f16[i] = __float2half(value[i]);
#else
            value_f16_nT[i] = __float2half(value[i]);
#endif
        }

#ifdef FA_KV_BLOCK_256
        // transpose value
        for(int h = 0; h < num_heads; h++) {
            for(int c = 0; c < head_dim; c++) {
                for(int r = 0; r < kv_size; r++) {
                    value_f16[h*kv_size*head_dim + c*kv_size + r] = __float2half(value[h*kv_size*head_dim + r*head_dim + c]);
                }
            }
        }
#endif

        hipStream_t stream;
        hipStreamCreate(&stream);
        float* d_query_f32;

        half *d_query, *d_key, *d_value, *d_value_nT, *d_mask, *d_padded_mask;
        float *d_score, *d_qkv;

        hipMalloc((void **)&d_query,   head_dim           * num_heads * sizeof(half));
        hipMalloc((void **)&d_query_f32,  head_dim        * num_heads * sizeof(float));

        hipMalloc((void **)&d_key,     head_dim * kv_size * num_heads * sizeof(half));
        hipMalloc((void **)&d_value,   head_dim * kv_size * num_heads * sizeof(half));
        hipMalloc((void **)&d_value_nT,  head_dim * kv_size * num_heads * sizeof(half));
        hipMalloc((void **)&d_mask,    kv_size * sizeof(half));
        hipMalloc((void **)&d_padded_mask,  16 *  kv_size * sizeof(half));

        hipMalloc((void **)&d_score,              kv_size * num_heads * sizeof(float));
        hipMalloc((void **)&d_qkv,     head_dim           * num_heads * sizeof(float));

        // copy CPU data to GPU memory blocks
        hipMemcpyAsync(d_query, query_f16, head_dim           * num_heads * sizeof(half), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_query_f32, query, head_dim           * num_heads * sizeof(float), hipMemcpyHostToDevice, stream);

        hipMemcpyAsync(d_key,   key_f16,   head_dim * kv_size * num_heads * sizeof(half), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_value, value_f16, head_dim * kv_size * num_heads * sizeof(half), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_value_nT, value_f16_nT, head_dim * kv_size * num_heads * sizeof(half), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_mask,  mask_f16,  kv_size * sizeof(half), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_padded_mask,  mask_f16_padded, 16 * kv_size * sizeof(half), hipMemcpyHostToDevice, stream);

        constexpr int kv_per_block = 256;
        constexpr int num_warps = 8;

        // assert(kv_size % kv_per_block == 0);
        dim3 grid_dim(kv_size / kv_per_block, num_heads, 1);
        dim3 block_dim(WARP_SIZE, num_warps, 1);

        int shmem =
            head_dim*2*sizeof(half) /* query buffer */ +
            (kv_per_block + 2)*sizeof(float) /* scores buffer */ +
            num_warps * (256 + 2) * sizeof(float) /* tensor core result buffer per warp */;
        printf("\n\nShared memory: %.2f KB\n\n", shmem/1024.0f);

        // print_array("CUDA key", key_f16, 4, 4, head_dim);

        for(int i = 0; i < head_dim * kv_size * num_heads; i ++) {
            key_f16[i] = __float2half(0.0f);
        }

        int reduce_block = ((grid_dim.x + WMMA_M - 1) / WMMA_M) * WMMA_N;
        printf("reduce block: %d\n", reduce_block);

        // half* d_red_buffer;
        // hipMalloc((void **)&d_red_buffer, head_dim * reduce_block * num_heads * sizeof(half));

        hipStreamSynchronize(stream);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, stream);
        bool paralell_kv = false;

        if(paralell_kv) {
            flash_attn<128, num_warps, 2, kv_per_block><<<grid_dim, block_dim, shmem, stream>>>(d_query, d_key, d_value, d_mask, kv_size, scale, reduce_block, head_dim*kv_size);

            fa_reduce<128, num_warps><<<num_heads, block_dim, shmem, stream>>>(
                d_key, d_qkv, kv_size, kv_size / kv_per_block, reduce_block);
        } else {
            // launch llama.cpp implementation
            const int nwarps = 8;
            constexpr int nqpb = 16;
            constexpr int ncpw = 128;
            printf("n_warps = %i\n", nwarps);

            dim3 blocks_num(1, num_heads, 1);
            dim3 block_dim(32, nwarps, 1);

            const size_t shmem_f_ = 16*(head_dim + nwarps*(ncpw + nqpb))*(sizeof(float)/2);

            flash_attn_ext_f16<128, nqpb, ncpw><<<blocks_num, block_dim, shmem_f_, stream>>>(
                (const char*)d_query_f32, (const char*)d_key, (const char*)d_value_nT, (const char*)d_padded_mask, d_qkv, scale,
                head_dim, 1, num_heads, 1, head_dim, kv_size, num_heads, 1, kv_size, 16*2,
                head_dim*4, head_dim*4, head_dim*num_heads*4,
                head_dim*2, head_dim*kv_size*2, head_dim*kv_size*num_heads*2,
                head_dim, num_heads, 1, 1);
        }

        hipEventRecord(stop, stream);
        hipEventSynchronize(stop);

        float millis = 0.0f;
        hipEventElapsedTime(&millis, start, stop);

        printf("cuda time: %.4f ms\n", millis);

        // transfer data from device to host
        hipMemcpyAsync(qkv_cuda, d_qkv, head_dim           * num_heads * sizeof(float), hipMemcpyDeviceToHost, stream);

        // half* red_buffer = (half*)malloc(head_dim * reduce_block * num_heads * sizeof(half));
        // hipMemcpyAsync(red_buffer, d_red_buffer, head_dim * reduce_block * num_heads * sizeof(half), hipMemcpyDeviceToHost, stream);

        // hipMemcpyAsync(scores, d_score, kv_size * num_heads * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(key_f16, d_key, head_dim * kv_size * num_heads * sizeof(half), hipMemcpyDeviceToHost, stream);

        hipStreamSynchronize(stream);

        float max_diff = 0.0f;
        int head_idx = 0, dim_idx = 0;

        for(int h = 0; h < num_heads; h++) {
            for(int i = 0; i < head_dim; i++) {
                if(fabs(qkv[h*head_dim + i] - qkv_cuda[h*head_dim + i]) > max_diff) {
                    max_diff = fabs(qkv[h*head_dim + i] - qkv_cuda[h*head_dim + i]);
                    head_idx = h;
                    dim_idx = i;
                }
            }
        }
        printf("R (%.4f) CUDA(%.4f) diff: %.4f - head = %d, dim = %d\n", qkv[head_idx*head_dim + dim_idx], qkv_cuda[head_idx*head_dim + dim_idx], max_diff, head_idx, dim_idx);

        if(paralell_kv) {
            float res = 0.0f;
            for(int i = 0;i < reduce_block; i ++) {
                printf(i < grid_dim.x ? "%.4f " : "[%.4f] ", __half2float(key_f16[head_idx*head_dim*kv_size + dim_idx*reduce_block + i]));
                res += __half2float(key_f16[head_idx*head_dim*kv_size + dim_idx*reduce_block + i]);
            }
            printf(" = %.4f\n", res);
        }
        print_array("CUDA QKV", qkv_cuda, 2, 16, head_dim);

        // clean up device memory
        hipFree(d_query);
        hipFree(d_key);
        hipFree(d_value);
        hipFree(d_qkv);
        hipFree(d_score);
    }

    free(query);
    free(key);
    free(value);
    free(qkv);
    free(qkv_cuda);
    free(scores);
    return 0;
}